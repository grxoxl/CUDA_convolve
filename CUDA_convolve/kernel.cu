#include "hip/hip_runtime.h"
﻿/* Preprocessor directives */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>

#define N 1024*1024 // 
#define BLOCK_SIZE 512

__device__ int device_min(int a, int b) {
	return (a < b) ? a : b;
}

/* CUDA kernel for convolution */
__global__ void conv_kernel(float* input, float* kernel, float* output, int input_size, int kernel_size) {

	// Find the starting index and step size for the loops
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tstep = blockDim.x * gridDim.x;

	int output_length = input_size + kernel_size - 1;

	for (int i = tid; i < output_length; i += tstep) {

		float conv_sum = 0;

		int max_kernel_size = device_min(input_size, i);
        for (int j = 0; j < max_kernel_size; j++) {
            conv_sum += input[j] * kernel[i - j];
        }

		output[i] = conv_sum;
	}
}


int main() {
    const int inputLength = 10;
    const int kernelLength = 3;
    const int resultLength = inputLength + kernelLength - 1;

    float* inputArray = (float*)malloc(inputLength * sizeof(float));
    float* kernelArray = (float*)malloc(kernelLength * sizeof(float));
    float* resultArray = (float*)malloc(resultLength * sizeof(float));
    float* d_inputArray, * d_kernelArray, * d_resultArray;

    // Initialize input and kernel arrays with random values 
    srand(time(NULL));
    for (int i = 0; i < inputLength; ++i) {
        inputArray[i] = (float)rand() / RAND_MAX;
        if (i < kernelLength) {
            kernelArray[i] = (float)rand() / RAND_MAX;
        }
    }

    // Allocate memory on GPU 
    hipMalloc(&d_inputArray, inputLength * sizeof(float));
    hipMalloc(&d_kernelArray, kernelLength * sizeof(float));
    hipMalloc(&d_resultArray, resultLength * sizeof(float));

    // Copy data from host to device 
    hipMemcpy(d_inputArray, inputArray, inputLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernelArray, kernelArray, kernelLength * sizeof(float), hipMemcpyHostToDevice);

    // Perform convolution 
    conv_kernel(d_inputArray, d_kernelArray, d_resultArray, inputLength, kernelLength);

    // Copy result back to host 
    hipMemcpy(resultArray, d_resultArray, resultLength * sizeof(float), hipMemcpyDeviceToHost);

    // Output the result 
    printf("Input Array: ");
    for (int i = 0; i < inputLength; ++i) {
        printf("%f ", inputArray[i]);
    }
    printf("\n");

    printf("Kernel Array: ");
    for (int i = 0; i < kernelLength; ++i) {
        printf("%f ", kernelArray[i]);
    }
    printf("\n");

    printf("Result Array: ");
    for (int i = 0; i < resultLength; ++i) {
        printf("%f ", resultArray[i]);
    }
    printf("\n");

    // Free memory 
    free(inputArray);
    free(kernelArray);
    free(resultArray);
    hipFree(d_inputArray);
    hipFree(d_kernelArray);
    hipFree(d_resultArray);

    return 0;
}
